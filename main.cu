#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include "stb_image.h"
#include "stb_image_write.h"
#include "arraylist.h"
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <iostream>

#define PI 3.14159265
#define imgchannels 3

#define KERNEL_COLUMNS 5
#define KERNEL_ROWS 5
#define BLOCK_SIZE_2D 32
#define BLOCK_SIZE_3D 8
#define SHARED_MEM_WIDTH (BLOCK_SIZE_2D + KERNEL_COLUMNS)

using namespace std;
// __device__ ArrayList* rtable[180];
double* rtable[181];
int pointer[181];


void convert_to_grayscale(const unsigned char*inputImage, unsigned char * outputImageData, int height, int width, int channels)
{
    for (int i = 0; i < height; ++i)
    {
      for (int j = 0; j < width; ++j)
      {
        float R = (float) inputImage[(width * i + j)*channels + 0];
        float G = (float) inputImage[(width * i + j)*channels + 1];
        float B = (float) inputImage[(width * i + j)*channels + 2];
        float gray_val = 0.299*R + 0.587*G + 0.114*B;
        outputImageData[(width * i + j)] = gray_val;
      }
    } 
}

__global__ void grayscaleKernel(const unsigned char*inputImage, unsigned char * outputImageData, int height, int width, int channels)
{
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;

  if (i>=0 && i<height && j>=0 && j<width)
  {
    float R = (float) inputImage[(width * i + j)*channels + 0];
    float G = (float) inputImage[(width * i + j)*channels + 1];
    float B = (float) inputImage[(width * i + j)*channels + 2];
    float gray_val = 0.299*R + 0.587*G + 0.114*B;
    outputImageData[(width * i + j)] = gray_val;
  }
}

__global__ void thresholdKernel(const unsigned char*InputImageData, unsigned char*OutputImageData, int height, int width, int channels, int threshold)
{
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;

  if (i>=0 && i<height && j>=0 && j<width)
  {
    for(int k=0; k<channels; k++)
    {
      float val = (float) InputImageData[(width * i + j)*channels + k];
      if(val>=threshold)
      {
        OutputImageData[(width * i + j)*channels + k] = 255;
      }
      else
      {
        OutputImageData[(width * i + j)*channels + k] = 0;
      }
    }
  }
}

__global__ void sobelKernel(const unsigned char*InputImageData, unsigned char * outputImageData, unsigned int * outputGradient, int height, int width)
{
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int x = blockIdx.x * blockDim.x + threadIdx.x;

  int allHorizontalX[2] = {-1, 1}; 
  int allHorizontalY[3] = {-1, 0, 1};
  int allCoefficientsX[6] = {-1, -2, -1, 1, 2, 1};
    
  int allVerticalX[3] = {-1, 0, 1}; 
  int allVerticalY[2] = {-1, 1};
  int allCoefficientsY[6] = {1, -1, 2, -2, 1, -1};

  if(x>=0 && x<width && y>=0 && y<height)
    {
        float h = 0;
        int k = 0;
        for(int i=0;i<2;i+=1)
        {
          for(int j=0;j<3;j+=1)
          {
            int index = (y + allHorizontalY[j]) * width + (x + allHorizontalX[i]);
            h += allCoefficientsX[k] * InputImageData[index];
            k+=1;
          }
        }

        float v = 0;
        k = 0;
        for(int i=0;i<3;i+=1)
        {
          for(int j=0;j<2;j+=1)
          {
            int index = (y + allVerticalY[j]) * width + (x + allVerticalX[i]);
            v += allCoefficientsY[k] * InputImageData[index];
            k+=1;
          }
        }

        h /= 5;
        v /= 5;
        float val = (float)sqrt((h*h) + (v*v));
        float gradient = atan(v / (h+0.000001));
        
        outputImageData[y*width+x] = val;
        outputGradient[y*width+x] = (gradient * 180 / PI) + 90;
    }
}

__global__ void convKernel(unsigned char*inputImage, float * kernel, unsigned char * outputImageData, int image_height, int image_width) {
    __shared__ float shared_mem[SHARED_MEM_WIDTH][SHARED_MEM_WIDTH];
    __shared__ float shared_kernel[KERNEL_ROWS * KERNEL_COLUMNS];

    int half_kernel = (int) (KERNEL_ROWS - 1) / 2;

    if (threadIdx.x == 0 and threadIdx.y == 0) {
        for (int i = 0; i < KERNEL_COLUMNS * KERNEL_ROWS; ++i)
            shared_kernel[i] = kernel[i];   
        int block_coord_x = blockDim.x * blockIdx.x;
        int block_coord_y = blockDim.y * blockIdx.y;
        int src_x, src_y;
        for (int i=0; i<SHARED_MEM_WIDTH; i++) {
            for (int j=0; j<SHARED_MEM_WIDTH; j++) {
                src_x = block_coord_x - half_kernel + i;
                src_y = block_coord_y - half_kernel + j;
                if(src_y >= 0 && src_y < image_height && src_x >=0 && src_x < image_width) {
                    shared_mem[j][i] = inputImage[src_x + image_width * src_y];
                } else {
                    shared_mem[j][i] = 0;
                }
            }
        }
    }

    __syncthreads();

    float sum = 0;
    int y, x;

    for (y= 0; y < KERNEL_COLUMNS; y++)
        for(x = 0; x<KERNEL_ROWS; x++)
            sum += shared_mem[threadIdx.y + y][threadIdx.x + x] * shared_kernel[y * KERNEL_COLUMNS + x];

    y = blockIdx.y * blockDim.y + threadIdx.y;
    x = blockIdx.x * blockDim.x + threadIdx.x;

    if(y < image_height && x < image_width)
        outputImageData[y * image_width + x] = sum;

}

float * get_avg_kernel() {
  float kernel_data[KERNEL_ROWS*KERNEL_COLUMNS];
  for(int i=0; i< KERNEL_ROWS*KERNEL_COLUMNS; i++){
      kernel_data[i] = 1.0/(KERNEL_ROWS*KERNEL_COLUMNS);
  }
  float * h_kernel_data = (float *) malloc(KERNEL_ROWS*KERNEL_COLUMNS*sizeof(float));
  memcpy(h_kernel_data, kernel_data, KERNEL_ROWS*KERNEL_COLUMNS*sizeof(float));
  return h_kernel_data;
}

__global__ void rtable_init_kernel(double** rtable, int *pointer, int initSize) {

  int tidx = threadIdx.x + blockDim.x * blockIdx.x;
  rtable[tidx] = (double*) malloc(initSize * sizeof(double));
  memset(rtable[tidx], 0, initSize * sizeof(double));
  pointer[tidx] = 0;
}

__global__ void rtable_create_kernel(double**rtable, int* pointer, unsigned char * template_edge, unsigned int * template_orientation, int height, int width, double centroid_x, double centroid_y) {
  int tidx = threadIdx.x + blockDim.x * blockIdx.x;
  unsigned int orientation = template_orientation[tidx];
  unsigned int edge = template_edge[tidx];
  if (edge != 0) {
    rtable[orientation][pointer[orientation]] = (tidx % width) - centroid_x;
    rtable[orientation][pointer[orientation]+1] = (tidx / width) - centroid_y;
    pointer[orientation]+=2;
  }
}

__global__ void rtable_print_kernel(double** rtable, int* pointer) {
  int tidx = threadIdx.x + blockDim.x * blockIdx.x;
  for(int index =0; index<pointer[tidx]/2; index++) {
    printf("(%.3f, %.3f) ", rtable[tidx][2*index], rtable[tidx][2*index+1]);
  }
  printf("\n");
  }

__global__ void accumulator_general(unsigned char* image, int * accum, int height, int width, int num_angle, int* pointer, double** rtable) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int theta = blockIdx.z * blockDim.z + threadIdx.z;

  if(image[y*width+x] != 0) {
    double *list = rtable[theta];
    for(int index =0; index<pointer[theta]/2; index++) {
      float vec_x = list[2*index];
      float vec_y = list[2*index+1];
      int x0 = (int) (x + vec_x);
      int y0 = (int) (y + vec_y);
      if (x0 >= 0 and x0 < width && y0 >=0 && y0 < height) {
        atomicAdd(&accum[(y0*width+x0)*num_angle+theta], 1);
      }
    }
  }
}


__global__ void accumulator_circle(unsigned char* image, int * accum, int height, int width, int num_radii) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int r = blockIdx.z * blockDim.z + threadIdx.z;

  if(image[y*width+x] != 0) {
    for(int theta =0; theta<360; theta++) {
      float r_cos_theta = r * cos((float) theta * PI/180);
      float r_sin_theta = r * sin((float) theta * PI/180);
      int x0 = (int) (x - r_cos_theta);
      int y0 = (int) (y - r_sin_theta);
      if (x0 >= 0 and x0 < width && y0 >=0 && y0 < height) {
        atomicAdd(&accum[(y0*width+x0)*num_radii+r], 1);
      }
    }
  }
}

unsigned char* get_image_accum_radii(int * accum, int height, int width, int radii, int num_radii) {
  unsigned char * image = (unsigned char *) malloc(width*height*sizeof(unsigned char));
  for (int y=0; y<height; y++) {
    for (int x=0; x<width; x++) {
      image[y*width+x] = (unsigned char) accum[(y*width+x)*num_radii+radii];
    }
  }
  return image;
}

__global__ void threshold(int * accum, int height, int width, int num_radii, int threshold_val) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int r = blockIdx.z * blockDim.z + threadIdx.z;

  if(accum[(y*width+x)*num_radii+r] < threshold_val) {
    accum[(y*width+x)*num_radii+r] = 0;
  }
}

int main()
{

  hipEvent_t start_kernel, stop_kernel;
  hipEventCreate(&start_kernel);
  hipEventCreate(&stop_kernel);


  int width, height, bpp;

  string filename = "input3.png";
  string outname = "output/hi.png";
  unsigned char *img;
  const unsigned char* image = stbi_load(filename.c_str(), &width, &height, &bpp, imgchannels);
  img = (unsigned char*)malloc(width*height*sizeof(unsigned char));
  cout << height << " " << width << " " << bpp << endl;
  convert_to_grayscale(image, img, height, width, imgchannels);
  stbi_write_png(outname.c_str(), width, height, 1, img, 0);

  /**********************CUDA-CODE*************************************/
  string gpu_out_edge = "output/hi_gpu_edge.png";
  string gpu_out_edge_thresholded = "output/hi_gpu_edge_thresholded.png";
  // string gpu_out_orientation = "output/hi_gpu_orientation.png";
  string gpu_blurred_image = "output/hi_gpu_blurred_image.png";
  string gpu_accum_image = "output/hi_gpu_accum_image.png";

  unsigned char * d_input;
  hipMalloc(&d_input, width*height*imgchannels*sizeof(unsigned char));
  hipMemcpy(d_input, image, width*height*imgchannels*sizeof(unsigned char), hipMemcpyHostToDevice);

  unsigned char * d_output;
  hipMalloc(&d_output, width*height*sizeof(unsigned char));

  unsigned char * d_output_edge;
  hipMalloc(&d_output_edge, width*height*sizeof(unsigned char));

  unsigned char * d_output_edge_thresholded;
  hipMalloc(&d_output_edge_thresholded, width*height*sizeof(unsigned char));

  unsigned int * d_output_orientation;
  hipMalloc(&d_output_orientation, width*height*sizeof(unsigned int));

  unsigned char * d_blurred_image;
  hipMalloc(&d_blurred_image, width*height*sizeof(unsigned char));

  float * d_kernel_data;
  hipMalloc(&d_kernel_data, KERNEL_ROWS*KERNEL_COLUMNS*sizeof(float));
  hipMemcpy(d_kernel_data, get_avg_kernel(), KERNEL_ROWS*KERNEL_COLUMNS*sizeof(float), hipMemcpyHostToDevice);

  const dim3 blockSize(BLOCK_SIZE_2D, BLOCK_SIZE_2D, 1);
  const int bx = ceil( (float) width/BLOCK_SIZE_2D);
  const int by = ceil( (float) height/BLOCK_SIZE_2D);
  const dim3 gridSize = dim3(bx, by);

  grayscaleKernel<<<gridSize, blockSize>>>(d_input, d_output, height, width, imgchannels);
  convKernel<<<gridSize, blockSize>>>(d_output, d_kernel_data, d_blurred_image, height, width);
  sobelKernel<<<gridSize, blockSize>>>(d_blurred_image, d_output_edge, d_output_orientation, height, width);
  thresholdKernel<<<gridSize, blockSize>>>(d_output_edge, d_output_edge_thresholded, height, width, 1, 10);

  int num_radii = 100;

  const dim3 blockSize_accum(BLOCK_SIZE_3D, BLOCK_SIZE_3D, BLOCK_SIZE_3D);
  const int bx_accum = ceil( (float) width/BLOCK_SIZE_3D);
  const int by_accum = ceil( (float) height/BLOCK_SIZE_3D);
  const int bz_accum = ceil( (float) num_radii/BLOCK_SIZE_3D);
  const dim3 gridSize_accum = dim3(bx_accum, by_accum, bz_accum);

  int * d_accum;
  hipMalloc(&d_accum, width*height*num_radii*sizeof(int));  
  hipMemset(d_accum, 0, width*height*num_radii*sizeof(int));

  // ----------------------------------------------------------------------------------------------------

  printf("1\n");
  double** d_rtable;
  int* d_pointer;
  int num_angles = 181;

  hipMalloc(&d_rtable, num_angles*sizeof(double*));
  hipMalloc(&d_pointer, num_angles*sizeof(int));
  rtable_init_kernel<<<1, num_angles>>>(d_rtable, d_pointer, 1000);
  rtable_create_kernel<<<gridSize, blockSize>>>(d_rtable, d_pointer, d_output_edge, d_output_orientation, height, width, width/2, height/2);
  hipDeviceSynchronize();

  int * d_accum_general;  
  hipMalloc(&d_accum_general, width*height*num_angles*sizeof(int));  
  hipMemset(d_accum_general, 0, width*height*num_angles*sizeof(int));
  hipEventRecord(start_kernel);
  accumulator_general<<<gridSize_accum, blockSize_accum>>>(d_output_edge_thresholded, d_accum_general, height, width, 181, d_pointer, d_rtable);
  hipDeviceSynchronize();
  hipEventRecord(stop_kernel);
  
  printf("2\n");

  hipEventSynchronize(stop_kernel);
  float k_time ;
  hipEventElapsedTime(&k_time, start_kernel, stop_kernel);
  cout << "GPU Time: " << k_time << " milliseconds" << endl;
  // ----------------------------------------------------------------------------------------------------

  accumulator_circle<<<gridSize_accum, blockSize_accum>>>(d_output_edge_thresholded, d_accum, height, width, num_radii);
  threshold<<<gridSize_accum, blockSize_accum>>>(d_accum_general, height, width, num_radii, 20);

  unsigned char * h_blurred_image = (unsigned char *) malloc(width*height*sizeof(unsigned char));
  hipMemcpy(h_blurred_image, d_blurred_image, width*height*sizeof(unsigned char), hipMemcpyDeviceToHost);

  unsigned char * h_output_edge = (unsigned char *) malloc(width*height*sizeof(unsigned char));
  hipMemcpy(h_output_edge, d_output_edge, width*height*sizeof(unsigned char), hipMemcpyDeviceToHost);

  unsigned char * h_output_edge_thresholded = (unsigned char *) malloc(width*height*sizeof(unsigned char));
  hipMemcpy(h_output_edge_thresholded, d_output_edge_thresholded, width*height*sizeof(unsigned char), hipMemcpyDeviceToHost);

  unsigned int * h_output_orientation = (unsigned int *) malloc(width*height*sizeof(unsigned int));
  hipMemcpy(h_output_orientation, d_output_orientation, width*height*sizeof(unsigned int), hipMemcpyDeviceToHost);

  int * h_accum = (int *) malloc(width*height*num_radii*sizeof(int));
  hipMemcpy(h_accum, d_accum, width*height*num_radii*sizeof(int), hipMemcpyDeviceToHost);

  unsigned char * accum_image = get_image_accum_radii( h_accum, height, width, 25, num_radii);

  stbi_write_png(gpu_blurred_image.c_str(), width, height, 1, h_blurred_image, 0);
  stbi_write_png(gpu_out_edge.c_str(), width, height, 1, h_output_edge, 0);
  stbi_write_png(gpu_out_edge_thresholded.c_str(), width, height, 1, h_output_edge_thresholded, 0);
  // stbi_write_png(gpu_out_orientation.c_str(), width, height, 1, h_output_orientation, 0);
  stbi_write_png(gpu_accum_image.c_str(), width, height, 1, accum_image, 0);

  hipFree(d_input);
  hipFree(d_output);
  hipFree(d_output_edge);
  hipFree(d_output_orientation);

  free(h_output_edge);
  free(h_output_orientation);
  /**********************CUDA-CODE-ENDS*************************************/

  return 0;
}
